#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include "cuda_raycaster.cuh"

__device__ uint32_t get_color_rgb_norm(float r, float g, float b) {
    return static_cast<uint32_t>(std::round(r * 255.f)) << 24 |
           static_cast<uint32_t>(std::round(g * 255.f)) << 16 |
           static_cast<uint32_t>(std::round(b * 255.f)) << 8 |
           0xFF;
}

__device__ uint32_t get_color_rgb(uint8_t r, uint8_t g, uint8_t b) {
    return static_cast<uint32_t>(r) << 24 |
           static_cast<uint32_t>(g) << 16 |
           static_cast<uint32_t>(b) << 8 |
           0xFF;
}

// Returns parametric function argument (t), returns -1.f in the case there was no hit
__device__ float get_sphere_hit(glm::vec3 center, float radius, glm::vec3 ray_origin, glm::vec3 ray_dir, float min) {
    glm::vec3 co = ray_origin - center;

    // Quadratic equation
    float a = glm::dot(ray_dir, ray_dir);
    float b = 2.f * glm::dot(ray_dir, co);
    float c = glm::dot(co, co) - radius * radius;

    float delta = b * b - 4 * a * c;

    if (delta < 0.f) {
        return -1.f;
    }

    // The ray is in enter state for that sphere
    float t = (-b - std::sqrt(delta)) / (2.f * a);
    if (t > min) {
        return t;
    }

    // The ray is in exit state for that sphere
    t = (-b + std::sqrt(delta)) / (2.f * a);
    if (t > min) {
        return t;
    }

    // The ray has missed the sphere
    return -1.f;
}

__device__ uint32_t on_hit(glm::vec3 hit_point, glm::vec3 normal, glm::vec3 color) {
    glm::vec3 light_pos = glm::vec3(0.f, 1.f, 0.f);

    // normal = 0.5f * (normal + 1.f);
    // return get_color_rgb_norm(normal.r, normal.g, normal.b);

    glm::vec3 light_dir = normalize(light_pos - hit_point);

    glm::vec3 res_color = color * glm::clamp(glm::dot(normal, light_dir), 0.f, 1.f);

    return get_color_rgb_norm(res_color.r, res_color.g, res_color.b);
}

__device__ uint32_t on_miss() {
    // Background color
    return get_color_rgb(60, 60, 60);
}

void check_cuda_error(const hipError_t &cuda_status, const char *msg) {
    if (cuda_status != hipSuccess) {
        std::cerr << msg << hipGetErrorString(cuda_status) << std::endl;
        std::terminate();
    }
}
__global__ void init(uint32_t *canvas, glm::vec3 *origins, glm::vec3 *dirs, int count) {
    uint32_t k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k < count) {
        canvas[k] = 0x000000FF;
        origins[k] = glm::vec3(0.f);
        dirs[k] = glm::vec3(0.f);
    }
}

cuda_raycaster::GPURayCaster::GPURayCaster(const csg::CSGTree& tree, int width, int height)
: m_width(width), m_height(height) {
    hipError_t cuda_status;

    // Allocate memory on the device using hipMalloc
    cuda_status = hipMalloc((void**)&m_dev_canvas, width * height * sizeof(uint32_t));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_origins, width * height * sizeof(glm::vec3));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_dirs, width * height * sizeof(glm::vec3));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");

    size_t threads_per_block = 1024;
    size_t blocks_num = m_width * m_height / threads_per_block + 1;
    init<<<blocks_num, threads_per_block>>>(
            m_dev_canvas,
            m_dev_origins,
            m_dev_dirs,
            width * height
    );
    hipDeviceSynchronize();

    cuda_status = hipMalloc((void**)&m_dev_radiuses, tree.get_sphere_count() * sizeof(float));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_centers, tree.get_sphere_count() * sizeof(glm::vec3));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_colors, tree.get_sphere_count() * sizeof(glm::vec3));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_node_array, tree.get_nodes_count() * sizeof(csg::Node));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");

    // Move tree data to the gpu
    cuda_status = hipMemcpy(m_dev_radiuses, tree.sphere_radiuses().data(), tree.get_sphere_count() * sizeof(float), hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(m_dev_centers, tree.sphere_centers().data(), tree.get_sphere_count() * sizeof(glm::vec3), hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(m_dev_colors, tree.sphere_colors().data(), tree.get_sphere_count() * sizeof(glm::vec3), hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(m_dev_node_array, tree.nodes().data(), tree.get_nodes_count() * sizeof(csg::Node), hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");

    m_spheres_count = tree.get_sphere_count();
    m_nodes_count = tree.get_nodes_count();
}

cuda_raycaster::GPURayCaster::~GPURayCaster() {
    hipFree(m_dev_origins);
    hipFree(m_dev_dirs);
    hipFree(m_dev_canvas);

    hipFree(m_dev_radiuses);
    hipFree(m_dev_centers);
    hipFree(m_dev_colors);
    hipFree(m_dev_node_array);
}

__global__ void find_dirs(
        glm::vec2 canvas_size,
        glm::vec3 eye,
        glm::mat4 inv_proj,
        glm::mat4 inv_view,
        glm::vec3* origins,
        glm::vec3* dirs,
        int width,
        int height
) {
    uint32_t k = blockIdx.x * blockDim.x + threadIdx.x;
    int count = width * height;
    if (k < count) {
        uint32_t x = k % width;
        uint32_t y = k / width;
        glm::vec2 viewport_coords = { static_cast<float>(x) / canvas_size.x, (static_cast<float>(y)) / canvas_size.y };
        viewport_coords = viewport_coords * 2.0f - 1.0f;

        // pixel's position in the world space
        glm::vec4 target = inv_proj * glm::vec4(viewport_coords.x, viewport_coords.y, -1.f, 1.f);
        dirs[k] = glm::vec3(inv_view * glm::vec4(normalize(glm::vec3(target) / target.w), 0.f)); // world space
        origins[k] = eye;
    }
}

__global__ void trace_ray(
        float *radiuses,
        glm::vec3 *centers,
        glm::vec3 *colors,
        uint32_t spheres_count,
        glm::vec3 *origins,
        glm::vec3 *dirs,
        uint32_t *canvas,
        int width,
        int height
) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k >= width * height) {
        return;
    }

    float t_min = FLT_MAX;
    int closest_sphere = -1;
    for (int i = 0; i < spheres_count; ++i) {
        float t = get_sphere_hit(centers[i], radiuses[i], origins[k], dirs[k], 0.f);

        if (t > 0.f && t < t_min) {
            t_min = t;
            closest_sphere = i;
        }
    }

    if (closest_sphere != -1) {
        glm::vec3 closest_hit = origins[k] + dirs[k] * t_min;
        canvas[k] = on_hit(closest_hit, normalize(closest_hit - centers[closest_sphere]),
                           colors[closest_sphere]);
    } else {
        canvas[k] = on_miss();
    }
}

void cuda_raycaster::GPURayCaster::update_canvas(renderer::Image &canvas,
                                                 const cuda_raycaster::GPURayCaster::Input &input) {
    resize(canvas.get_width(), canvas.get_height());
    size_t threads_per_block = 1024;
    size_t blocks_num = m_width * m_height / threads_per_block + 1;

    // First kernel: find rays and origins
    find_dirs<<<blocks_num, threads_per_block>>>(
            input.canvas,
            input.eye,
            input.inv_proj,
            input.inv_view,
            m_dev_origins,
            m_dev_dirs,
            m_width,
            m_height
    );
    hipDeviceSynchronize();

    // Second kernel: ray casting
    trace_ray<<<blocks_num, threads_per_block>>>(
            m_dev_radiuses,
            m_dev_centers,
            m_dev_colors,
            m_spheres_count,
            m_dev_origins,
            m_dev_dirs,
            m_dev_canvas,
            m_width,
            m_height
    );
    hipDeviceSynchronize();

    hipError_t cuda_status;
    cuda_status = hipMemcpy((void*)canvas.raw(), m_dev_canvas, m_width * m_height * sizeof(uint32_t), hipMemcpyDeviceToHost);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
}

void cuda_raycaster::GPURayCaster::resize(int width, int height) {
    if (width == m_width && height == m_height) {
        return;
    }
    m_width = width;
    m_height = height;

    hipFree(m_dev_dirs);
    hipFree(m_dev_origins);
    hipFree(m_dev_canvas);

    hipError_t cuda_status;
    cuda_status = hipMalloc((void**)&m_dev_origins, width * height * sizeof(glm::vec3));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_dirs, width * height * sizeof(glm::vec3));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_canvas, width * height * sizeof(uint32_t));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
}

