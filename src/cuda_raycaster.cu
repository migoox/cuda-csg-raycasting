#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include "cuda_raycaster.cuh"

__device__ uint32_t get_color_rgb_norm(float r, float g, float b) {
    return static_cast<uint32_t>(std::round(r * 255.f)) << 24 |
           static_cast<uint32_t>(std::round(g * 255.f)) << 16 |
           static_cast<uint32_t>(std::round(b * 255.f)) << 8 |
           0xFF;
}

__device__ uint32_t get_color_rgb(uint8_t r, uint8_t g, uint8_t b) {
    return static_cast<uint32_t>(r) << 24 |
           static_cast<uint32_t>(g) << 16 |
           static_cast<uint32_t>(b) << 8 |
           0xFF;
}

// Returns parametric function argument (t), returns -1.f in the case there was no hit
__device__ float get_sphere_hit(glm::vec3 center, float radius, glm::vec3 ray_origin, glm::vec3 ray_dir, float min) {
    glm::vec3 co = ray_origin - center;

    // Quadratic equation
    float a = glm::dot(ray_dir, ray_dir);
    float b = 2.f * glm::dot(ray_dir, co);
    float c = glm::dot(co, co) - radius * radius;

    float delta = b * b - 4 * a * c;

    if (delta < 0.f) {
        return -1.f;
    }

    // The ray is in enter state for that sphere
    float t = (-b - std::sqrt(delta)) / (2.f * a);
    if (t > min) {
        return t;
    }

    // The ray is in exit state for that sphere
    t = (-b + std::sqrt(delta)) / (2.f * a);
    if (t > min) {
        return t;
    }

    // The ray has missed the sphere
    return -1.f;
}

__device__ uint32_t on_hit(glm::vec3 hit_point, glm::vec3 normal, glm::vec3 color) {
    glm::vec3 light_pos = glm::vec3(0.f, 1.f, 0.f);

    // normal = 0.5f * (normal + 1.f);
    // return get_color_rgb_norm(normal.r, normal.g, normal.b);

    glm::vec3 light_dir = normalize(light_pos - hit_point);

    glm::vec3 res_color = color * glm::clamp(glm::dot(normal, light_dir), 0.f, 1.f);

    return get_color_rgb_norm(res_color.r, res_color.g, res_color.b);
}

__device__ uint32_t on_miss() {
    // Background color
    return get_color_rgb(60, 60, 60);
}

void check_cuda_error(const hipError_t &cuda_status, const char *msg) {
    if (cuda_status != hipSuccess) {
        std::cerr << msg << hipGetErrorString(cuda_status) << std::endl;
        std::terminate();
    }
}
__global__ void init(uint32_t *canvas, glm::vec3 *origins, glm::vec3 *dirs, int count) {
    uint32_t k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k < count) {
        canvas[k] = 0x000000FF;
        origins[k] = glm::vec3(0.f);
        dirs[k] = glm::vec3(0.f);
    }
}

cuda_raycaster::GPURayCaster::GPURayCaster(const csg::CSGTree& tree, int width, int height)
: m_width(width), m_height(height) {
    hipError_t cuda_status;

    // Allocate memory on the device using hipMalloc
    cuda_status = hipMalloc((void**)&m_dev_canvas, width * height * sizeof(uint32_t));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_origins, width * height * sizeof(glm::vec3));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_dirs, width * height * sizeof(glm::vec3));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");

    size_t threads_per_block = 1024;
    size_t blocks_num = m_width * m_height / threads_per_block + 1;
    init<<<blocks_num, threads_per_block>>>(
            m_dev_canvas,
            m_dev_origins,
            m_dev_dirs,
            width * height
    );
    hipDeviceSynchronize();

    cuda_status = hipMalloc((void**)&m_dev_radiuses, tree.get_sphere_count() * sizeof(float));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_centers, tree.get_sphere_count() * sizeof(glm::vec3));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_colors, tree.get_sphere_count() * sizeof(glm::vec3));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_node_array, tree.get_nodes_count() * sizeof(csg::Node));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");

    // Move tree data to the gpu
    cuda_status = hipMemcpy(m_dev_radiuses, tree.sphere_radiuses().data(), tree.get_sphere_count() * sizeof(float), hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(m_dev_centers, tree.sphere_centers().data(), tree.get_sphere_count() * sizeof(glm::vec3), hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(m_dev_colors, tree.sphere_colors().data(), tree.get_sphere_count() * sizeof(glm::vec3), hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
    cuda_status = hipMemcpy(m_dev_node_array, tree.nodes().data(), tree.get_nodes_count() * sizeof(csg::Node), hipMemcpyHostToDevice);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");

    m_spheres_count = tree.get_sphere_count();
    m_nodes_count = tree.get_nodes_count();
}


cuda_raycaster::GPURayCaster::~GPURayCaster() {
    hipFree(m_dev_origins);
    hipFree(m_dev_dirs);
    hipFree(m_dev_canvas);

    hipFree(m_dev_radiuses);
    hipFree(m_dev_centers);
    hipFree(m_dev_colors);
    hipFree(m_dev_node_array);
}

__global__ void find_dirs(
        glm::vec2 canvas_size,
        glm::vec3 eye,
        glm::mat4 inv_proj,
        glm::mat4 inv_view,
        glm::vec3* origins,
        glm::vec3* dirs,
        int width,
        int height
) {
    uint32_t k = blockIdx.x * blockDim.x + threadIdx.x;
    int count = width * height;
    if (k < count) {
        uint32_t x = k % width;
        uint32_t y = k / width;
        glm::vec2 viewport_coords = { static_cast<float>(x) / canvas_size.x, (static_cast<float>(y)) / canvas_size.y };
        viewport_coords = viewport_coords * 2.0f - 1.0f;

        // pixel's position in the world space
        glm::vec4 target = inv_proj * glm::vec4(viewport_coords.x, viewport_coords.y, -1.f, 1.f);
        dirs[k] = glm::vec3(inv_view * glm::vec4(normalize(glm::vec3(target) / target.w), 0.f)); // world space
        origins[k] = eye;
    }
}

__device__ csg::PointState csg_point_classify(float t, glm::vec3 normal, glm::vec3 ray_dir) {
    if (t == 0.f) {
        return csg::PointState::Miss;
    }

    if (dot(normal, ray_dir) > 0.f) {
        return csg::PointState::Exit;
    }

    if (dot(normal, ray_dir) < 0.f) {
        return csg::PointState::Enter;
    }

    return csg::PointState::Miss;
}

__device__ csg::IntersectionResult csg_intersect(
        csg::Node *nodes,
        int prim_count,
        int nodes_count,
        float *radiuses,
        glm::vec3 *centers,
        glm::vec3 *colors,
        glm::vec3 origin,
        glm::vec3 dir,
        csg::Node node,
        float min
) {
    // Stop condition
    if (node.type == csg::Node::Type::Sphere) {
        float t = get_sphere_hit(centers[node.prim_id], radiuses[node.prim_id], origin, dir, min);
        return csg::IntersectionResult {
                t,
                t == -1.f ? glm::vec3(0.f) : normalize(origin + t * dir - centers[node.prim_id]),
                node.id
        };
    }

    float min_l = min;
    float min_r = min;

    // Recursive call
    csg::IntersectionResult res_l = csg_intersect(nodes, prim_count, nodes_count, radiuses, centers, colors, origin, dir, nodes[node.get_left_id()], min_l);
    csg::IntersectionResult res_r = csg_intersect(nodes, prim_count, nodes_count, radiuses, centers, colors, origin, dir, nodes[node.get_right_id()], min_r);

    csg::PointState state_l = csg_point_classify(res_l.t, res_l.normal, dir);
    csg::PointState state_r = csg_point_classify(res_r.t, res_r.normal, dir);
    while (true) {
        csg::CSGActions actions = csg::CSGActions(state_l, state_r, node);
        if (actions.has_action(csg::CSGActions::Miss)) {
            return csg::IntersectionResult { -1.f, glm::vec3(0.f), -1 }; // Miss
        }

        if (actions.has_action(csg::CSGActions::RetLeft) ||
            (actions.has_action(csg::CSGActions::RetLeftIfCloser) && res_l.t <= res_r.t)) {
            return res_l;
        }

        if (actions.has_action(csg::CSGActions::RetRight) ||
            (actions.has_action(csg::CSGActions::RetRightIfCloser) && res_r.t <= res_l.t)) {
            if (actions.has_action(csg::CSGActions::FlipRight)) {
                return csg::IntersectionResult { res_r.t, -res_r.normal, res_r.leaf_id };
            }
            return res_r;
        }

        if (actions.has_action(csg::CSGActions::LoopLeft) ||
            (actions.has_action(csg::CSGActions::LoopLeftIfCloser) && res_l.t <= res_r.t)) {
            min_l = res_l.t;
            res_l = csg_intersect(nodes, prim_count, nodes_count, radiuses, centers, colors, origin, dir, nodes[node.get_left_id()], min_l);
            state_l = csg_point_classify(res_l.t, res_l.normal, dir);
        } else if (actions.has_action(csg::CSGActions::LoopRight) ||
                   (actions.has_action(csg::CSGActions::LoopRightIfCloser) && res_r.t <= res_l.t)) {
            min_r = res_r.t;
            res_r = csg_intersect(nodes, prim_count, nodes_count, radiuses, centers, colors, origin, dir, nodes[node.get_right_id()], min_r);
            state_r = csg_point_classify(res_r.t, res_r.normal, dir);
        } else {
            return csg::IntersectionResult { -1.f, glm::vec3(0.f), -1 }; // Miss
        }
    }
}

__global__ void csg_trace_ray(
        csg::Node *nodes,
        float *radiuses,
        glm::vec3 *centers,
        glm::vec3 *colors,
        int prim_count,
        int nodes_count,
        uint32_t *canvas,
        int width,
        int height,
        glm::vec3 *origins,
        glm::vec3 *dirs
) {
    uint32_t k = blockIdx.x * blockDim.x + threadIdx.x;
    int count = width * height;

    if (k >= count) {
        return;
    }

    if (nodes_count <= 1) {
        canvas[k] = on_miss();
    }

    auto result = csg_intersect(nodes, prim_count, nodes_count, radiuses, centers, colors, origins[k], dirs[k], nodes[1], 0.f);

    if (result.leaf_id == -1) {
        canvas[k] = on_miss();
    } else {
        canvas[k] = on_hit(origins[k] + dirs[k] * result.t, result.normal, colors[nodes[result.leaf_id].prim_id]);
    }
}

__global__ void trace_ray(
        float *radiuses,
        glm::vec3 *centers,
        glm::vec3 *colors,
        uint32_t spheres_count,
        glm::vec3 *origins,
        glm::vec3 *dirs,
        uint32_t *canvas,
        int width,
        int height
) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k >= width * height) {
        return;
    }

    float t_min = FLT_MAX;
    int closest_sphere = -1;
    for (int i = 0; i < spheres_count; ++i) {
        float t = get_sphere_hit(centers[i], radiuses[i], origins[k], dirs[k], 0.f);

        if (t > 0.f && t < t_min) {
            t_min = t;
            closest_sphere = i;
        }
    }

    if (closest_sphere != -1) {
        glm::vec3 closest_hit = origins[k] + dirs[k] * t_min;
        canvas[k] = on_hit(closest_hit, normalize(closest_hit - centers[closest_sphere]),
                           colors[closest_sphere]);
    } else {
        canvas[k] = on_miss();
    }
}

void cuda_raycaster::GPURayCaster::update_canvas(renderer::Image &canvas,
                                                 const cuda_raycaster::GPURayCaster::Input &input) {
    resize(canvas.get_width(), canvas.get_height());
    size_t threads_per_block = 1024;
    size_t blocks_num = m_width * m_height / threads_per_block + 1;

    // First kernel: find rays and origins
    find_dirs<<<blocks_num, threads_per_block>>>(
            input.canvas,
            input.eye,
            input.inv_proj,
            input.inv_view,
            m_dev_origins,
            m_dev_dirs,
            m_width,
            m_height
    );
    hipDeviceSynchronize();

    // Second kernel: ray casting
    if (input.show_csg) {

        csg_trace_ray<<<blocks_num, threads_per_block>>>(
                m_dev_node_array,
                m_dev_radiuses,
                m_dev_centers,
                m_dev_colors,
                m_spheres_count,
                m_nodes_count,
                m_dev_canvas,
                m_width,
                m_height,
                m_dev_origins,
                m_dev_dirs
        );
    } else {
        trace_ray<<<blocks_num, threads_per_block>>>(
                m_dev_radiuses,
                m_dev_centers,
                m_dev_colors,
                m_spheres_count,
                m_dev_origins,
                m_dev_dirs,
                m_dev_canvas,
                m_width,
                m_height
        );
    }
    hipDeviceSynchronize();

    hipError_t cuda_status;
    cuda_status = hipMemcpy((void*)canvas.raw(), m_dev_canvas, m_width * m_height * sizeof(uint32_t), hipMemcpyDeviceToHost);
    check_cuda_error(cuda_status, "[CUDA]: hipMemcpy failed: ");
}

void cuda_raycaster::GPURayCaster::resize(int width, int height) {
    if (width == m_width && height == m_height) {
        return;
    }
    m_width = width;
    m_height = height;

    hipFree(m_dev_dirs);
    hipFree(m_dev_origins);
    hipFree(m_dev_canvas);

    hipError_t cuda_status;
    cuda_status = hipMalloc((void**)&m_dev_origins, width * height * sizeof(glm::vec3));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_dirs, width * height * sizeof(glm::vec3));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
    cuda_status = hipMalloc((void**)&m_dev_canvas, width * height * sizeof(uint32_t));
    check_cuda_error(cuda_status, "[CUDA]: hipMalloc failed: ");
}

